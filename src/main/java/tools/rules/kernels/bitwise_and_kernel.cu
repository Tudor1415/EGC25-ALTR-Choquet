
#include <hip/hip_runtime.h>
// bitwise_and_kernel.cu
extern "C"
__global__ void computeAndOperation(int* cover1, int* cover2, int* result, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result[i] = cover1[i] & cover2[i];
    }
}
