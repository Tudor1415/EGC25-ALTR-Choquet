#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/set_operations.h>

extern "C"
void setIntersectionThrust(
    const int* indices1, int nnz1,
    const int* indices2, int nnz2,
    int* resultIndices, int* resultSize)
{
    thrust::device_ptr<const int> d_indices1(indices1);
    thrust::device_ptr<const int> d_indices2(indices2);
    thrust::device_ptr<int> d_resultIndices(resultIndices);

    thrust::device_vector<int> vec1(d_indices1, d_indices1 + nnz1);
    thrust::device_vector<int> vec2(d_indices2, d_indices2 + nnz2);

    thrust::device_vector<int> resultVec(std::min(nnz1, nnz2));

    auto end = thrust::set_intersection(
        vec1.begin(), vec1.end(),
        vec2.begin(), vec2.end(),
        resultVec.begin());

    int numElements = end - resultVec.begin();
    *resultSize = numElements;

    // Copy result to output array
    thrust::copy(resultVec.begin(), end, d_resultIndices);
}
